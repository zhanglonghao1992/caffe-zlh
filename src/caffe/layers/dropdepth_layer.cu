#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
#include <time.h>
#include <stdio.h>

#include "caffe/layers/dropdepth_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void DropdepthForward(const int n, const Dtype* in,
    unsigned int* mask, const unsigned int threshold, Dtype* out, unsigned int* d) {
  CUDA_KERNEL_LOOP(index, n) {
    mask[index] = *d;
    out[index] = in[index] * (mask[index] < threshold);
  }
}

template <typename Dtype>
void DropdepthLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  if (this->phase_ == TRAIN) {
    unsigned int* mask = static_cast<unsigned int*>(rand_vec_.mutable_gpu_data());
    
    //unsigned int* b = new unsigned int[1];
    unsigned int b = caffe_rng_rand();
    //caffe_gpu_rng_uniform(1,b);
    //LOG(INFO)<<"b:"<<b[0];
    //LOG(INFO)<<"b:"<<b;
    //unsigned int* c;
    unsigned int* c;
    //hipMalloc((void**)&c,1*sizeof(unsigned int));
    //hipMemcpy(c,&b,1*sizeof(unsigned int),hipMemcpyHostToDevice);
    hipMalloc((void**)&c,1*sizeof(unsigned int));
    hipMemcpy(c,&b,1*sizeof(unsigned int),hipMemcpyHostToDevice);
    
    //unsigned int a;
    //hipMemcpy(&a,c,1*sizeof(unsigned int),hipMemcpyDeviceToHost); 
    //LOG(INFO)<<"a:"<<a;

    //LOG(INFO)<<"UINT_MAX:"<<UINT_MAX;  

    //for (int i = 0; i < count; ++i){
        //mask[i] = *c;                                
    //}
    //LOG(INFO)<<"5";
    // set thresholds
    // NOLINT_NEXT_LINE(whitespace/operators)
    DropdepthForward<Dtype><<<CAFFE_GET_BLOCKS(count),
      CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, mask, int_thres_, top_data, c);
    //LOG(INFO)<<"6";
    //delete[] b;
    hipFree(c);
    //LOG(INFO)<<"7";  
    CUDA_POST_KERNEL_CHECK;
    //LOG(INFO)<<"8";
  } else {
    caffe_copy(count, bottom_data, top_data);
  }
}

template <typename Dtype>
__global__ void DropdepthBackward(const int n, const Dtype* in_diff,
    const unsigned int* mask, const unsigned int threshold, Dtype* out_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = in_diff[index] * (mask[index] < threshold);
  }
}

template <typename Dtype>
void DropdepthLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    if (this->phase_ == TRAIN) {
      const unsigned int* mask =
          static_cast<const unsigned int*>(rand_vec_.gpu_data());
      const int count = bottom[0]->count();
      
      // NOLINT_NEXT_LINE(whitespace/operators)
      DropdepthBackward<Dtype><<<CAFFE_GET_BLOCKS(count),
        CAFFE_CUDA_NUM_THREADS>>>(
         count, top_diff, mask, int_thres_, bottom_diff);
      
      CUDA_POST_KERNEL_CHECK;
    } else {
      caffe_copy(top[0]->count(), top_diff, bottom_diff);
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(DropdepthLayer);

}  // namespace caffe
